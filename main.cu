#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#define BLOCK_NUM 32
#define THREAD_NUM 256

bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}

__global__ static void HelloCUDA(char* result, int num)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	
	char p_HelloCUDA[] = "Hello CUDA!";
	for(int i = bid*THREAD_NUM + tid; i < num; i+=BLOCK_NUM*THREAD_NUM) {
		result[i] = p_HelloCUDA[i];
	}
}

int main(int argc,char *argv[])
{
	if (!InitCUDA())
	{
		return 0;
	}
	
	char *device_result = 0;
	char host_result[12] = {0};
	
	CUDA_SAFE_CALL(hipMalloc((void **) &device_result,sizeof(char) *11));
	
	unsigned int timer = 0;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));
	
	HelloCUDA<<<BLOCK_NUM,THREAD_NUM,0>>>(device_result,11);
	CUT_CHECK_ERROR("Kernel execution failed\n");
	
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	CUT_SAFE_CALL( cutStopTimer( timer));
	printf("Processing time: %f (ms)\n", cutGetTimerValue( timer));
	CUT_SAFE_CALL( cutDeleteTimer( timer));

	CUDA_SAFE_CALL( hipMemcpy(&host_result, device_result, sizeof(char) * 11, hipMemcpyDeviceToHost));
	printf("%s\n", host_result);

	CUDA_SAFE_CALL( hipFree(device_result));
	CUT_EXIT(argc, argv);	
	
	return 0;
}